#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#define CheckErrorUtil(err) CheckError(err, __FUNCTION__, __LINE__)
#define CheckErrorMsgUtil(err, msg) CheckErrorMsg(err, msg, __FUNCTION__, __LINE__)

inline void CheckError(hipError_t const err, char const* const fun, const int line)
{
    if (err)
    {
        printf("CUDA Error Code[%d]: %s\n%s() Line:%d\n", err, hipGetErrorString(err), fun, line);
        exit(1);
    }
}

inline void CheckErrorMsg(hipError_t const err, char const* const msg, char const* const fun, int const line)
{
    if (err)
    {
        printf("CUDA Error Code[%d]: %s\n%s() Line:%d\n%s\n", err, hipGetErrorString(err), fun, line, msg);
        exit(1);
    }
}

void GenerateTestArrays(int const N, float* const a, float* const b, float* const c, float* const ref);
void CompareArrays(int const N, float const* const a, float const* const b);

__global__ void ArraysSum(float* const a, float* const b, float* const c, int const N)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    if (i < N)
        c[i] = a[i] + b[i];
}

int main()
{
    dim3 gridSize;
    dim3 blockSize;

    int    const N       = 2053;
    size_t const N_BYTES = N * sizeof(float);
    int const BLOCK_SIZE = 512; 
   
    float *aH, *bH, *cH, *refH;
    float *aD, *bD, *cD;

    aH = (float*)malloc(N_BYTES);
    bH = (float*)malloc(N_BYTES);
    cH = (float*)malloc(N_BYTES);
    refH = (float*)malloc(N_BYTES);

    GenerateTestArrays(N, aH, bH, cH, refH);

    CheckErrorUtil(hipMalloc((void**)&aD, N_BYTES));
    CheckErrorUtil(hipMalloc((void**)&bD, N_BYTES));
    CheckErrorUtil(hipMalloc((void**)&cD, N_BYTES));

    CheckErrorUtil(hipMemcpy(aD, aH, N_BYTES, hipMemcpyHostToDevice));
    CheckErrorUtil(hipMemcpy(bD, bH, N_BYTES, hipMemcpyHostToDevice));

    blockSize.x = BLOCK_SIZE; blockSize.y = 1; blockSize.z = 1;
    gridSize.x = ((N + BLOCK_SIZE - 1) / BLOCK_SIZE); gridSize.y = 1; gridSize.z = 1;

    ArraysSum<<<gridSize, blockSize>>>(aD, bD, cD, N);

    // Get Errors from kernel
    CheckErrorUtil(hipDeviceSynchronize());
    CheckErrorUtil(hipGetLastError());

    CheckErrorUtil(hipMemcpy(cH, cD, N_BYTES, hipMemcpyDeviceToHost));

    CompareArrays(N, cH, refH);

    CheckError(hipFree(aD));
    CheckError(hipFree(bD));
    CheckError(hipFree(cD));

    free(aH); free(bH); free(cH); free(refH);
    
    CheckError(hipDeviceReset());
    return 0;
}

void GenerateTestArrays(int const N, float* const a, float* const b, float* const c, float* const ref)
{
    int i;

    srand((unsigned)time(NULL));

    for(i = 0; i < N; i++)
    {
        a[i] = (float)rand();
        b[i] = (float)rand();
        c[i] = 0.0f;
        ref[i] = a[i] + b[i];
    }
}

void CompareArrays(int const N, float const* const a, float const* const b)
{
    int i;
    int different = 0;

    for(i = 0; i < N; i++)
    {
        different = (a[i] != b[i]);
        if(different)
            break;
    }

    if(different)
    {
        printf("Arrays do not match.\n");
    }
    else
    {
        printf("Arrays match.\n");
    }
}
